#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ int myStrlen(char arr[]){ 
	int counter=0;
	for(int i=0;i<7;i++){// max size in Semiconservative group is 5
		if(arr[i]=='\0')
			return counter;
		else
			counter++;
	}
	return counter;
}

//Computing values and overrides vec A
__global__ void findMuTantCuda(char *d_A, char *seq1,int offset,int isMax,int seq2Size,char *d_Seq2Temp,int *d_weigh)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index<seq2Size){// need only strlen(size2) trheads to do the job
    	if(isMax==1){ //its maximum
    		int flag1 = 0, flag2 = 0, i, j;
			char ConservativeGroup[][5] = { { 'S','T','A','\0' } ,{ 'N','E','Q','K','\0' } ,{ 'N','D','E','Q','\0'},
			{ 'N','H','Q','K','\0' },{ 'Q','H','R','K','\0' },{ 'M','I','L','V','\0' },{ 'M','I','L','F','\0' },{ 'H','Y','\0' },{ 'F','Y','W','\0'} };
		
			for (i = 0; i < 9; i++) { //checking if the letters are in the same conservative group
				int size = myStrlen(ConservativeGroup[i]);// checking the size of the next group of chars
				for (j = 0; j < size; j++) {
			
					if (seq1[index+offset] == ConservativeGroup[i][j])//first char found in a group of close attributes
						flag1 = 1;
					if (d_A[index] == ConservativeGroup[i][j])//second char found in a group of close attributes
						flag2 = 1;
					if (flag1 == 1 && flag2 == 1)// the chars are in the same conservative group
					{
						d_Seq2Temp[index]=d_A[index];
						return;
					}
				}
				flag1 = 0;
				flag2 = 0;
				size = 0;
			}
			d_Seq2Temp[index]=seq1[index+offset]; //if not in the same conservative group return the letter from seq1
    	
    	 }
    	 else{//its minimum
    	 	if(d_weigh[2]>=d_weigh[3]){ //try to get the max weight to get minimum result
    	 	    char semiConservativeGroup[11][7] = { { 'C','S','A','\0' } ,{ 'A','T','V','\0' } ,{ 'S','A','G','\0'},
				{ 'S','T','N','K','\0'},{ 'S','T','P','A','\0'},{ 'S','G','N','D','\0'},{ 'S','N','D','E','Q','K','\0' },{ 'N','D','E','Q','H','K','\0'}
				,{'N','E','Q','H','R','K','\0'},{'F','V','L','I','M','\0' },{'H','F','Y','\0' } };
				for (int i = 0; i < 9; i++) { //checking if the letters are in the same Semiconservative group
					int size = myStrlen(semiConservativeGroup[i]);// checking the size of the next group of chars
					for (int j = 0; j < size; j++) {
						if (seq1[index+offset] == semiConservativeGroup[i][j])//first char found in a Semiconservative group
						{
							if(j==0)
								d_Seq2Temp[index]= semiConservativeGroup[i][1];
							else
								d_Seq2Temp[index]= semiConservativeGroup[i][0];
							return;
						}

					}
					size = 0;
				}
				
    	 	 }

			if(seq1[index+offset]!='B') // if the char cant be fount in any Semiconservative group then return letter that cant be found in any group
				d_Seq2Temp[index]= 'B';
			else
				d_Seq2Temp[index]= 'J';
		
    	 }
    	 
    }

}

int computeOnGPU(char *seq1,char *seq2,int numOfElementSeq1,int numOfElementsSeq2,int offset,int isMax,int *weight)
{

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    size_t sizeSeq1 =numOfElementSeq1 * sizeof(char);
    size_t sizeSeq2 = numOfElementsSeq2 * sizeof(char);
    size_t sizeWeight=4*sizeof(int);
  


    char *d_Seq2;
    char *d_Seq1;
    char *d_Seq2Temp;
    int *d_weight;
    err = hipMalloc((void **)&d_Seq2, sizeSeq2);     // Allocate memory on GPU to copy seq1 from the host
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_Seq1, sizeSeq1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        err = hipMalloc((void **)&d_Seq2Temp, sizeSeq2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        err = hipMalloc((void **)&d_weight, sizeWeight);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Seq2, seq2, sizeSeq2, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        // Copy data from host to the GPU memory
    err = hipMemcpy(d_Seq1, seq1, sizeSeq1, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        err = hipMemcpy(d_weight, weight, sizeWeight, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numOfElementsSeq2 + threadsPerBlock - 1) / threadsPerBlock;
    findMuTantCuda<<<blocksPerGrid, threadsPerBlock>>>(d_Seq2, d_Seq1,offset,isMax,numOfElementsSeq2,d_Seq2Temp,d_weight);
    
    
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(seq2, d_Seq2Temp, sizeSeq2, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_Seq2) != hipSuccess)
    {
       fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        // Free allocated memory on GPU
    if (hipFree(d_Seq2Temp) != hipSuccess)
    {
       fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        // Free allocated memory on GPU
    if (hipFree(d_Seq1) != hipSuccess)
    {
       fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
        if (hipFree(d_weight) != hipSuccess)
    {
       fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}
